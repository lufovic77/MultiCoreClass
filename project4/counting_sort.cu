#include <hip/hip_runtime.h>
// 나중에 지우기
#include <iostream>
#include <vector>
#include <algorithm>
using namespace std;

__host__ void counting_sort(int arr[], int size, int max_val)
{
   int i, j;
   // fill in 
   int histogram[max_val];

   for (i=0; i<max_val; i++){
      histogram[i] = 0;
   }
   for (i=0; i<size; i++){
      histogram[arr[i]]++;
   }
   int idx = 0;
   for (i=0;i<max_val;i++){
      for (j=0;j<histogram[i];j++){
            arr[idx++] = i;
      }
   }
}
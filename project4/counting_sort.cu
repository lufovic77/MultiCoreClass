#include <hip/hip_runtime.h>
// 나중에 지우기
#include <iostream>
#include <vector>
#include <algorithm>
#include <cstdio>
using namespace std;

__global__ void countingSortKernel(int * histogram_d , int * array_d, int max_val_h, int size, int num_blocks, int num_threads){
   int idx = blockIdx.x * blockDim.x + threadIdx.x;
   int total_threads = num_blocks*num_threads;
   int interval = size / total_threads;
   int start = idx*interval, end;
   if(blockIdx.x == num_blocks-1 && threadIdx.x == num_threads-1)
      end = size;
   else 
      end = start+interval;
   __syncthreads();
   for(int i=start;i<end;i++)
      atomicAdd(&histogram_d[array_d[i]], 1);
   __syncthreads();
}

__host__ void counting_sort(int arr[], int size, int max_val)
{
   //int i, j;
   int num_blocks = 3, num_threads = 10;
   // fill in 
   int arrSize = size*sizeof(int), histoSize = max_val*sizeof(int);
   int * histogram_d;
   int * histogram;
   int * arr_d;
   histogram = new int[histoSize];
   // Initializing histogram
   hipMalloc(&histogram_d, histoSize);
	hipMemset(histogram_d, 0, histoSize);
   hipMalloc(&arr_d, arrSize);
   hipMemcpy(arr_d, arr, arrSize, hipMemcpyHostToDevice);

   countingSortKernel <<<num_blocks, num_threads>>> (histogram_d, arr_d, max_val, size, num_blocks, num_threads);
   hipMemcpy(histogram, histogram_d, histoSize, hipMemcpyDeviceToHost);
   int idx = 0;
   // copy the output result
   for (int i=0;i<max_val;i++){
      for (int j=0;j<histogram[i];j++){
            arr[idx++] = i;
      }
   }
   hipFree(histogram);
   hipFree(histogram_d);
   hipFree(arr_d);
}